
#include "hip/hip_runtime.h"

#include "hipfft/hipfft.h"
#include "hipfft/hipfftw.h"
#include <stdlib.h>

#include<math.h>  
#include "hipfft/hipfftXt.h"
#include <stdio.h>
#include<iostream>
#include<stdlib.h>
#include <fstream>

//using namespace std;
//#define PI 3.1415926535897932384626433832795028841971               //定义圆周率值
//#define FFT_N 1024	
//#define N 1024   //通道数*点数   一个点的电压值占用4个字节 
//#define M 4
//#define fs 48000
//#define d 0.06
//#define v 340
////int fs,d,v,FFT_N;
///*fs=48000;%采样频率
////M=4;%麦克风阵元数 
////d=0.06;%阵元间距
////FFT_N=768;%采样点数
////v=340;%空气中声速
////f_xx每路数据FFT之后的数据 */
////struct complex EE(struct complex a,struct complex b);//对两个复数进行乘法运算
////void FFT(struct complex *xin);//对输入的复数组进行快速傅里叶变换（FFT）
//int DOA(struct complex *f_x1,struct complex *f_x2,struct complex *f_x3,struct complex *f_x4);//波束形成计算角度 
//float a_x1[N],a_x2[N],a_x3[N],a_x4[N];
//long b_x1[N],b_x2[N],b_x3[N],b_x4[N];
//FILE *fp1 = NULL;
//struct complex x1[FFT_N],x2[FFT_N],x3[FFT_N],x4[FFT_N];
// struct complex Energy;
//     struct complex Y,Ytemp;
//  struct complex t[4];
//     struct complex cov_x[4][1024],temp[4][1024],f_x[4][1024];
//	 float delay,DOA_scan[181]={0},Earray[181][1024],maxx=0;
//     int angle=0;
//	 int doa_scan[181];
extern "C" int runtest(long *adbuffer)
{

	//long ADData;
	//float fVolt;
	//for (int z = 0; z < N; z++)
	//{
	//	b_x1[z]=adbuffer[4*z];
	//	b_x2[z]=adbuffer[4*z+1];
	//	b_x3[z]=adbuffer[4*z+2];
	//	b_x4[z]=adbuffer[4*z+3];


	//}
	//for (int j = 0; j < N; j++)
	//{
	//	a_x1[j]=(20000.0/16384)*((b_x1[j]^0x2000)&0x3FFF)- 10000.0;
	//	a_x2[j]=(20000.0/16384)*((b_x2[j]^0x2000)&0x3FFF)- 10000.0;
	//	a_x3[j]=(20000.0/16384)*((b_x3[j]^0x2000)&0x3FFF)- 10000.0;
	//	a_x4[j]=(20000.0/16384)*((b_x4[j]^0x2000)&0x3FFF)- 10000.0;
	//}
	////if (true)
	////{
	////	return 1000;
	////}
	////ADData=(adbuffer[i]^0x2000)&0x3FFF;//电压默认-10v---+10v
	////	fVolt=((20000.0/16384) * ADData - 10000.0)/1000;


	// int i=0;int deg=0;	float end_point_power=0.0;
 //    struct complex s[FFT_N];          //FFT输入和输出：从S[1]开始存放，根据大小自己定义
 //    struct complex x1[FFT_N],x2[FFT_N],x3[FFT_N],x4[FFT_N];
 //    for(int k=0;k<FFT_N;k++)
 //       { 
 //       x1[k].real=a_x1[k]/1000 ; x1[k].imag=0;
	//	x2[k].real=a_x2[k]/1000  ; x2[k].imag=0;
 //       x3[k].real=a_x3[k]/1000  ; x3[k].imag=0;
 //       x4[k].real=a_x4[k]/1000   ; x4[k].imag=0;
 //       }   

	///* for(int u=0;u<FFT_N;u++)
	//	end_point_power+=abs(x1[u].real)*abs(x1[u].real);
	//if(end_point_power<1500){
	//	end_point_power=0;
	//	return 1000;}*/
 // /*********计算FFT*******/
 //  fft(FFT_N,x1);//for(int i=1;i<FFT_N+1)f_x1=x1;                                        //进行快速福利叶变换
 //  fft(FFT_N,x2);//f_x2=x2;  
 //  fft(FFT_N,x3);//f_x3=x3
 //  fft(FFT_N,x4);//f_x4=x4;                                     //目的是为了验证对FFT_N点信号进行FFT之后的效果比对                                //目的是为了验证对FFT_N点信号进行FFT之后的效果比对                
 //  deg=DOA(x1,x2,x3,x4);  
 //  return deg;
	return 0;










}
///*******************************************************************
//函数原型：struct complex EE(struct complex b1,struct complex b2)  
//函数功能：对两个复数进行乘法运算
//输入参数：两个以联合体定义的复数a,b
//输出参数：a和b的乘积，以联合体的形式输出
//*******************************************************************/
//struct complex EE(struct complex a,struct complex b)      
//{
// struct complex c;
// c.real=a.real*b.real-a.imag*b.imag;    
// c.imag=a.real*b.imag+a.imag*b.real;
// return(c);
//}
//
///*****************************************************************
//函数原型：void FFT(struct complex *xin,int FFT_N)
//函数功能：对输入的复数组进行快速傅里叶变换（FFT）
//输入参数：*xin复数结构体组的首地址指针，struct型
//*****************************************************************/
////void FFT(struct complex *xin)
////{
////  int f,m,nv2,nm1,i,k,l,j=0;
////  struct complex u,w,t;
////  static struct complex f_xin; 
////   nv2=FFT_N/2;                 //变址运算，即把自然顺序变成倒位序，采用雷德算法
////   nm1=FFT_N-1;  
////   for(i=0;i<nm1;i++)        
////   {
////    if(i<j)                    //如果i<j,即进行变址
////     {
////      t=xin[j];           
////      xin[j]=xin[i];
////      xin[i]=t;
////     }
////    k=nv2;                    //求j的下一个倒位序
////    while(k<=j)               //如果k<=j,表示j的最高位为1   
////     {           
////      j=j-k;                 //把最高位变成0
////      k=k/2;                 //k/2，比较次高位，依次类推，逐个比较，直到某个位为0
////     }
////   j=j+k;                   //把0改为1
////  }
////                         
////  {
////   int le,lei,ip;                            //FFT运算核，使用蝶形运算完成FFT运算
////    f=FFT_N;
////   for(l=1;(f=f/2)!=1;l++)                  //计算l的值，即计算蝶形级数
////           ;
////  for(m=1;m<=l;m++)                         // 控制蝶形结级数
////   {                                        //m表示第m级蝶形，l为蝶形级总数l=log（2）FFT_N
////    le=2<<(m-1);                            //le蝶形结距离，即第m级蝶形的蝶形结相距le点
////    lei=le/2;                               //同一蝶形结中参加运算的两点的距离
////    u.real=1.0;                             //u为蝶形结运算系数，初始值为1
////    u.imag=0.0;
////    w.real=cos(PI/lei);                     //w为系数商，即当前系数与前一个系数的商
////    w.imag=-sin(PI/lei);
////    for(j=0;j<=lei-1;j++)              //控制计算不同种蝶形结，即计算系数不同的蝶形结
////     {
////      for(i=j;i<=FFT_N-1;i=i+le)         //控制同一蝶形结运算，即计算系数相同蝶形结
////       {
////        ip=i+lei;                           //i，ip分别表示参加蝶形运算的两个节点
////        t=EE(xin[ip],u);                    //蝶形运算，详见公式
////        xin[ip].real=xin[i].real-t.real;
////        xin[ip].imag=xin[i].imag-t.imag;
////        xin[i].real=xin[i].real+t.real;
////        xin[i].imag=xin[i].imag+t.imag;
////       }
////      u=EE(u,w);                           //改变系数，进行下一个蝶形运算
////     }
////   }
////  }
////}
//
//int DOA(struct complex *f_x1,struct complex *f_x2,struct complex *f_x3,struct complex *f_x4)
//{
//	 for(int i=0;i<181;i++)
//		 doa_scan[i]=i-90;
//	 Y.real=0;Y.imag=0;
//     for(int i=0;i<FFT_N;i++)
//     {
//        f_x[0][i]=f_x1[i];
//        f_x[1][i]=f_x2[i];
//        f_x[2][i]=f_x3[i];
//        f_x[3][i]=f_x4[i];               
//     }                                 
//    
//for(int i=0;i<=180;i++)
//    for(int j=0;j<FFT_N;j++)
//    {    
//        //计算权系数t 
//		delay=2*PI*j*fs*d*sin(PI*doa_scan[i]/180)/v/FFT_N;
//         for(int k=0;k<M;k++)
//         {t[k].real=cos(k*delay);
//          t[k].imag=sin(k*delay);
//         }
//       //对fdata补偿得到Y
//         for(int k=0;k<M;k++) 
//		 {
//			 Ytemp=EE(t[k],f_x[k][j]);
//			 Y.real=Y.real+Ytemp.real;
//			 Y.imag=Y.imag+Ytemp.imag;
//		 }
//         Earray[i][j]=Y.real*Y.real+Y.imag*Y.imag;
//		  Y.real=0;Y.imag=0;
//    }
//
//     for(int i=0;i<=180;i++)
//     {
//       for(int j=0;j<FFT_N/2;j++)
//     DOA_scan[i]+=Earray[i][j];
//       if(DOA_scan[i]>maxx)
//       {maxx=DOA_scan[i];angle=i-90;}
//     }
//     return angle;          
//}
//
//
////           t[0].real=cos(delay);t[0].imag=sin(delay);
////           t[1].real=4*cos(delay);t[1].imag=4*sin(delay);
////           t[2].real=8*cos(delay);t[2].imag=8*sin(delay); 
////           t[3].real=16*cos(delay);t[3].imag=16*sin(delay);   
///*     for(int i=1;i<FFT_N+1;i++)
//     f_x[0][i]=f_x1[i];
//     for(int i=1;i<FFT_N+1;i++)
//     f_x[1][i]=f_x2[i];
//     for(int i=1;i<FFT_N+1;i++)
//     f_x[2][i]=f_x3[i];
//     for(int i=1;i<FFT_N+1;i++)
//     f_x[3][i]=f_x4[i];   */
// /*    for(int i=0;i<FFT_N;i++)
//     {
//          for(int ii=0;ii<M;ii++)                                              //计算协方差矩阵
//           for(int jj=0;jj<M;jj++)
//             cov_x[ii][jj]=EE(f_x[ii][i],f_x[jj][i]);                          //求每个频点对应的4路信号的协方差 
//      for(int j=-90;j<=90;j++)
//       {
//           delay=-2*PI*i*fs*d*sin(PI*j/180)/v/FFT_N;
//           for(int k=0;k<M;k++)
//           {t[k].real=k*cos(delay);
//            t[k].imag=k*sin(delay);
//           }
//                                                                                //计算每个频点，每个角度的能量值t‘*cov_x*t；
//           for(int ii=0;ii<M;i++)                                               //计算t'*cov_x =temp 
//            for(int jj=0;jj<M;jj++)
//               {
//                    temp[1][ii].real+=EE(t[jj],cov_x[jj][ii]).real;
//                    temp[1][ii].imag+=EE(t[jj],cov_x[jj][ii]).imag;
//               }
//          for(int ii=0;ii<M;ii++)
//               {
//                    Energy.real=EE(temp[1][ii],t[ii]).real; 
//                    Energy.imag=EE(temp[1][ii],t[ii]).imag; 
//               }                                           //计算temp*t 
//            Earray[j][i]=sqrt(Energy.real*Energy.real+Energy.imag*Energy.imag);   
//       }
//     } 
//     */




